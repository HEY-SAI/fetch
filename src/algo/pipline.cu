#include "hip/hip_runtime.h"


#if 1
#define ECHO(estr)   LOG("---\t%s\n",estr)
#else
#define ECHO(estr)
#endif

#define LOG(...)     printf(__VA_ARGS__)
#define REPORT(estr) LOG("%s(%d): %s()\n\t%s\n\tEvaluated to false.\n",__FILE__,__LINE__,__FUNCTION__,estr)
#define TRY(e)       do{ECHO(#e);if(!(e)){REPORT(#e);goto Error;}}while(0)

#define CUREPORT(ecode,estr) LOG("%s(%d): %s()\n\t%s\n\t%s\n",__FILE__,__LINE__,__FUNCTION__,estr,hipGetErrorString(ecode))
#define CUTRY(e)             do{ hipError_t ecode; ECHO(#e); ecode=(e); (if(ecode!=hipSuccess){CUREPORT(ecode,#e);goto Error;}}while(0)

#define NEW(T,e,N)   TRY((e)=(T*)malloc(sizeof(T)*(N)))
#define ZERO(T,e,N)  memset((e),0,sizeof(T)*(N))

#define countof(e)   (sizeof(e)/sizeof(*(e)))

// Original pipeline steps
// 1. pixel average (downsample)
// 2. frame average
// 3. cast to u16 (output pixel type)
// 4. invert
// 5. format frame (planewise...this will be implicit)
// 6. wrap
// 7. warp (lut)

/* PLAN
  INPUT
  * Count on 32MB/image (4 channels, no downsampling)
  * Expect 32 pixel aligned rows (16 after wrap)
      - one block will access BX*WORK columns

  1. VRAM based accumulator for frame averaging
     Just need the space for one extra image.
     Call to kernel increments accumulalator which will write an image
     to the destination buffer every k calls.

     * Accumulator is aligned to BX*WORK
  2. cast and invert are pixel-wise and can be done inline.  frame format is implicit
  3. wrap and unwarp are combined
    1. wrap
       Rows have an even number of pixels so turn will be aligned to a pixel border.  This
       makes this just a copy...might be able to do this in place.
    2. unwrap 
       This is a lut mapping...could introduce linear interp, though this could 
       be taken care of with the lut and the norm
    3. subsampling is done by lut

  THREADS PER BLOCK (BX,BY)
    - BX   should be 1 warp (32)
    - BY   should be set to run max threads per block (BY=1024/BX)
    - WORK should be just enough to hide memory latency
  BLOCKS
    X: ceil( width/BX/WORK )
    Y: ceil( height*nchan/BY)
*/

struct pipeline_t
{ float* accumulalator; ///< device memory, the size of one image.  Allocate with enough memory so rows are aligned to BX*WORK.
  float    norm;        ///< 1.0/the frame count as a float - set by launcher (eg. for ctx.every=4, this should be 0.25)
           m,b;         ///< slope and intercept for intensity scaling
  unsigned every,       ///< the number of frames to average
           count,       ///< the number of frames that have been pushed to the accumulator
           emit;        ///< true when (count+1)%every==0 - set by launcher
  unsigned turn;

  unsigned stride;      ///< number of elements between rows of input.  output stride will be stride/2.
  unsigned w,h;         ///< input width and height (height is nrows*nchan)
};

__constant__ unsigned *lut_;       ///< look up table for unwarp (ctx.w/2 number of elements).  Set by launch on first call or if width changes.
__constant__ float    *lut_norms_; ///< weight per pixel element

template<typename T,    ///< pixel type (input and output)
         unsigned BX,   ///< block size in X
         unsigned BY,   ///< block size in Y (channel dimension unrolled into Y)
         unsigned WORK  ///< number of elements to process per thread
         >
__global__ void __launch_bounds__(BX*BY,1) /* max threads, min blocks */
         pipeline_kernel(pipeline_t ctx, const T* __restrict__ src, T* __restrict__ dst)
{ __shared__ float buf[BY][WORK*BX+1];
  const int ox=threadIdx.x+blockIdx.x*WORK*BX,
            oy=threadIdx.y+blockIdx.y*BY;
  // --- Read ---
  if(oy>=h)
    return;

  src+=ox+oy*ctx.istride;
  if(blockIdx.x!=(gridDim.x-1))
  { 
    #pragma unroll
    for(int i=0;i<WORK;++i)
      buf[threadIdx.y][threadIdx.y+i*BX]=src[i*BX];
  } else // last block, need to check bounds
  { unsigned rem=(ctx.w-blockIdx.x*WORK*BX)/BX;  // divisible if data is aligned to BX
    for(int i=0;i<rem;++i)
      buf[threadIdx.y][threadIdx.x+i*BX]=src[i*BX];
  }
  __syncthreads();
  
  // --- Accumulate ---  
  if(ctx.every==1) // no frame averaging
  { 
    #pragma unroll
    for(int i=0;i<WORK;++i)                // put frame average in buffer
      buf[threadIdx.y][threadIdx.x+i*BX]=fma(buf[threadIdx.y][threadIdx.x+i*BX],m,b);
  } else {
    float *acc=ctx.acc+ox+oy*ctx.istride;
    #pragma unroll
    for(int i=0;i<WORK;++i)                // accumulate
      acc[i*BX]+=buf[threadIdx.y][threadIdx.x+i*BX];
    if(!ctx.emit)
      return;
    #pragma unroll
    for(int i=0;i<WORK;++i)                // put frame average in buffer
      buf[threadIdx.y][threadIdx.x+i*BX]=fma(acc[i*BX],m*ctx.norm,b);
  }

  // --- Wrap and Unwarp ---
  // *** FIXME - since this is an averaging step need this to go into a float buffer
  //             that gets mapped to dst later when it can be cast.
  // *** FIXME - fractional overlap of samples
  //           - can do this by tracking fractional part of lut and contributing
  //             to the lut[i] pixel and lut[i]+1'th pixel.
  //           - need to work out norms...norms has to be in dest space
  if(blockIdx.x < ctx.turn/(WORK*BX))      // --- block all forward scan
  { unsigned *lut   = lut_      +ox;
    float    *norms = lut_norms_+ox;
    dst+=oy*ctx.stride;                    // output to even lines (halved stride cancels with doubled line #)
    #pragma unroll
    for(int i=0;i<WORK;++i)
      dst[lut[i*BX]]+=buf[threadIdx.y][threadIdx.x+i*BX]*norms[i*BX];
  }
  else if(blockIdx.x > ctx.turn/(WORK*BX)) // --- block all reverse scan
  { unsigned *lut   = lut_      +ctx.turn-ox;
    float    *norms = lut_norms_+ctx.turn-ox;
    dst+=(2*oy+1)*ctx.stride/2;            // output to odd lines
    #pragma unroll
    for(int i=WORK-1;i>=0;--i)
      dst[lut[-i*BX]]+=buf[threadIdx.y][threadIdx.x+i*BX]*norms[-i*BX];    
  } else                                   // --- Block stradles turn
  {

  }
}


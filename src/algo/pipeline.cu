#include "hip/hip_runtime.h"
#include "pipeline.h"
#include "pipeline-image.h"
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdio.h>  //for printf
#include <stdlib.h> //for malloc
#include <string.h> //for memset

#include "hip/hip_runtime.h"

#define BX_   (32)
#define BY_   (32)
#define WORK_ (8)

#if 1
#define ECHO(estr)   LOG("---\t%s\n",estr)
#else
#define ECHO(estr)
#endif

#define LOG(...)     printf(__VA_ARGS__)
#define REPORT(estr) LOG("%s(%d): %s()\n\t%s\n\tEvaluated to false.\n",__FILE__,__LINE__,__FUNCTION__,estr)
#define TRY(e)       do{ECHO(#e);if(!(e)){REPORT(#e);goto Error;}}while(0)
#define NEW(T,e,N)   TRY((e)=(T*)malloc(sizeof(T)*(N)))
#define ZERO(T,e,N)  memset((e),0,sizeof(T)*(N))

#define CUREPORT(ecode,estr) LOG("%s(%d): %s()\n\t%s\n\t%s\n",__FILE__,__LINE__,__FUNCTION__,estr,hipGetErrorString(ecode))
#define CUTRY(e)             do{ hipError_t ecode; ECHO(#e); ecode=(e); if(ecode!=hipSuccess){CUREPORT(ecode,#e);goto Error;}}while(0)
#define CUWARN(e)            do{ hipError_t ecode; ECHO(#e); ecode=(e); if(ecode!=hipSuccess){CUREPORT(ecode,#e);           }}while(0)
#define CUNEW(T,e,N)    CUTRY(hipMalloc((void**)&(e),sizeof(T)*(N)))
#define CUZERO(T,e,N)   CUTRY(hipMemset((e),0,sizeof(T)*(N)))


#define countof(e)   (sizeof(e)/sizeof(*(e)))


/**
 * The parameter collection that gets passed to the kernel
 */
struct pipeline_ctx_t
{ float    * __restrict__ accumulator; ///< device memory, the size of one source image.  Allocate with enough memory so rows are aligned to BX*WORK.
  unsigned * __restrict__ lut;         ///< look up table for unwarp (ctx.w/2 number of elements).  Set by launch on first call or if width changes.
  float    * __restrict__ lut_fpart;   ///< fractional part of pixel position for lut.  If a smaple is mapped to 4.6, then this is 0.6.  0.4 of the sample will go to pixel 4, and 0.6 will go to pixel 5
  float    * __restrict__ lut_norms;   ///< weight contributed by an input sample to it's main pixel and the neighbor.  Should be interleaved.  Should sum to 1 for each output pixel.
  float    norm,        ///< 1.0/the frame count as a float - set by launcher (eg. for ctx.every=4, this should be 0.25)
           m,b;         ///< slope and intercept for intensity scaling
  unsigned istride,     ///< number of elements between rows of source.
           ostride;     ///< number of elements between rows of output.
  unsigned w,h;         ///< source width and height (height is nrows*nchan)
};

/**
 * The object that manages pipeline execution.
 */
typedef struct pipeline_t_
{ pipeline_ctx_t ctx;
  unsigned       count, ///< the number of frames that have been pushed to the accumulator
                 every; ///< the number of frames to average
  double         samples_per_scan;
  bool           invert;
  unsigned       downsample;
  unsigned       alignment;      ///< output rows are aligned to this target number of elements.
  void  * __restrict__ src,  ///< device buffer
        * __restrict__ dst;  ///< device buffer
  float * __restrict__ tmp;  ///< device buffer
} *pipeline_t;

//
// --- DEVICE ---
//

template<typename T,unsigned BX,unsigned BY,unsigned WORK>
__device__ void load(const pipeline_ctx_t & ctx, const T* __restrict__ src, float * __restrict__ buf[BY],const unsigned ox, const unsigned oy)
{
  src+=ox+oy*ctx.istride;
  if(blockIdx.x!=(gridDim.x-1))
  { 
    #pragma unroll
    for(int i=0;i<WORK;++i)
      buf[threadIdx.y][threadIdx.y+i*BX]=src[i*BX];
  } else                                              // last block, need to check bounds
  { unsigned rem=(ctx.w-blockIdx.x*WORK*BX)/BX;       // divisible if data is aligned to BX
    for(int i=0;i<rem;++i)
      buf[threadIdx.y][threadIdx.x+i*BX]=src[i*BX];
    __syncthreads();
  }
}

template<typename T,unsigned BX,unsigned BY,unsigned WORK,bool EMIT_FRAME_AVERAGE>
__device__ void accumulate(const pipeline_ctx_t & ctx,float* __restrict__ buf[BY],const unsigned ox, const unsigned oy)
{
  float *acc=ctx.accumulator+ox+oy*ctx.istride;
  #pragma unroll
  for(int i=0;i<WORK;++i)                // accumulate
    acc[i*BX]+=buf[threadIdx.y][threadIdx.x+i*BX];
  if(!EMIT_FRAME_AVERAGE)
    return;
  #pragma unroll
  for(int i=0;i<WORK;++i)                // put frame average in buffer
    buf[threadIdx.y][threadIdx.x+i*BX]=fma(acc[i*BX],ctx.m*ctx.norm,ctx.b);
  #pragma unroll
  for(int i=0;i<WORK;++i)                // reset accumulator
    acc[i*BX]=0;
}

/**
 * dst width should be aligned to BX*WORK
 * lut takes care of wrapping and turn and whatnot
 * BY must be greater than or equal to 3.
 * 
 * TODO(?): should buffer the load from shared memory for the lut,fpart and norms, maybe?
 */
template<typename T,unsigned BX,unsigned BY,unsigned WORK>
__device__ void warp(const pipeline_ctx_t & ctx, const float * __restrict__ buf[BY], float *__restrict__ dst,const unsigned ox, const unsigned oy)
{ unsigned *lut   = ctx.lut      +  ox;
  float    *fpart = ctx.lut_fpart+  ox;
  float    *norms = ctx.lut_norms+2*ox;
  dst+=2*oy*ctx.ostride;
  #pragma unroll
  for(int i=0;i<WORK;++i)
  { dst[lut[i*BX]  ]+=buf[threadIdx.y][threadIdx.x+i*BX]*norms[2*i*BX  ]*fpart[i*BX];
    dst[lut[i*BX]+1]+=buf[threadIdx.y][threadIdx.x+i*BX]*norms[2*i*BX+1]*(1.0f-fpart[i*BX]);
  }
}

//
// --- KERNELS ---
//


/**
 * \param dst should be allocated with one extra (trash) column a the end.
 *            If w,h,c are the width, height, and number of channels of 
 *            src then d should have dimensions (w/2+1,2*h,c).
 */
template<typename T,    ///< pixel type (input and output)
         unsigned BX,   ///< block size in X
         unsigned BY,   ///< block size in Y (channel dimension unrolled into Y)
         unsigned WORK, ///< number of elements to process per thread
         bool     ENABLE_FRAME_AVERAGE,
         bool     EMIT_FRAME_AVERAGE     ///< true when (count+1)%every==0 - set by launcher
         >
__global__ void __launch_bounds__(BX*BY,1) /* max threads, min blocks */
pipeline_kernel(pipeline_ctx_t ctx, const T* __restrict__ src, float* __restrict__ dst)
{ __shared__ float buf[BY][WORK*BX+1]; // for pre-warp
  const unsigned ox=threadIdx.x+blockIdx.x*WORK*BX,
                 oy=threadIdx.y+blockIdx.y*BY;
  if(oy>=ctx.h)
    return;
  load<T,BX,BY,WORK>(ctx,src,buf,ox,oy);
  if(ENABLE_FRAME_AVERAGE)
  { accumulate<T,BX,BY,WORK,EMIT_FRAME_AVERAGE>(ctx,buf,ox,oy);
    if(!EMIT_FRAME_AVERAGE)
      return;
  } else 
  { 
    #pragma unroll
    for(int i=0;i<WORK;++i) // in-place intensity scaling
      buf[threadIdx.y][threadIdx.x+i*BX]=fma(buf[threadIdx.y][threadIdx.x+i*BX],ctx.m,ctx.b);
  }
  warp<T,BX,BY,WORK>(ctx,buf,dst,ox,oy);
}

/**
 * Cast array from float to T.
 * src and dst should be the same shape but may be different types.
 * Both must have width aligned to BX*WORK
 * Both must have heigh aligned to BY
 */
template<typename T,    ///< pixel type (input and output)
         unsigned BX,   ///< block size in X
         unsigned BY,   ///< block size in Y (channel dimension unrolled into Y)
         unsigned WORK  ///< number of elements to process per thread
         >
__global__ void __launch_bounds__(BX*BY,1)
cast_kernel(T*__restrict__ dst, const float* __restrict__ src, unsigned stride)
{ __shared__ float buf[BY][WORK*BX+1];
  const int ox=threadIdx.x+blockIdx.x*WORK*BX,
            oy=threadIdx.y+blockIdx.y*BY;
  //if(oy>=h) return; // for unaligned y, uncomment and add an argument to the kernel call
  src+=ox+oy*stride;  
  dst+=ox+oy*stride;

  #pragma unroll
  for(int i=0;i<WORK;++i) //LOAD
    buf[threadIdx.y][threadIdx.y+i*BX]=src[i*BX];
  for(int i=0;i<WORK;++i) //STORE
    dst[i*BX]=(T)(buf[threadIdx.y][threadIdx.y+i*BX]);
}

//
// --- PUBLIC INTERFACE ---
//

pipeline_t pipeline_make(const pipeline_param_t *params)
{ pipeline_t self=NULL;
  TRY(params);
  NEW(pipeline_t_,self,1);
  ZERO(pipeline_t_,self,1);
  self->every            = params->frame_average_count;
  self->samples_per_scan = params->sample_rate_MHz*1.0e6/(double)params->scan_rate_Hz;
  self->invert           = params->invert_intensity;
  self->downsample       = params->pixel_average_count;
  self->alignment        = BX_*WORK_;
  self->ctx.norm         = 1.0f/(float)self->every;
  return self;
Error:
  return NULL;
}

void pipeline_free(pipeline_t *self)
{ if(self && *self)
  { void *ptrs[]={self[0]->ctx.accumulator,
                  self[0]->ctx.lut,
                  self[0]->ctx.lut_fpart,
                  self[0]->ctx.lut_norms,
                  self[0]->src,
                  self[0]->dst,
                  self[0]->tmp};
    for(int i=0;i<countof(ptrs);++i)
      if(ptrs[i])
        CUWARN(hipFree(ptrs[i]));
    free(*self); *self=NULL;
  }
}


#define EPS (1e-3)
static unsigned pipeline_get_output_width(pipeline_t self, const double inwidth)
{ const double d=inwidth/self->samples_per_scan; // 1 - duty
  //max derivative of the cosine warp adjusted to cos(2pi*(d/2)) is the zero point
  //and the positive part of the warp function goes from 0 to 1.
  const double maxslope=M_PI*(1.0-d)/inwidth/cos(M_PI*d);
  const double amplitude=1.0/maxslope;
  const unsigned w=self->alignment*(unsigned)(amplitude/(float)self->downsample/self->alignment);
  TRY(-EPS<d && d<=(0.5+EPS));
  TRY(0<w && w<inwidth);
  return w;
Error:
  return 0;
}
#undef EPS

int pipeline_get_output_dims(pipeline_t self, const pipeline_image_t src,unsigned *w, unsigned *h, unsigned *nchan)
{ TRY(self && src);
  if(nchan) *nchan=src->nchan;
  if(h)     *h=src->h*2;
  if(w)     TRY(*w=pipeline_get_output_width(self,src->w));
  return 1;
Error:
  return 0;
}

static int pipeline_alloc_accumulator(pipeline_t self, const pipeline_image_t src)
{ if(self->ctx.accumulator)
    hipFree(self->ctx.accumulator);
  const unsigned w     = self->alignment*((src->w+self->alignment-1)/self->alignment); // increase number of columns to align
  const unsigned nelem = w*src->h*src->nchan;
  CUNEW(float,self->ctx.accumulator,nelem);
  CUZERO(float,self->ctx.accumulator,nelem);
  return 1;
Error:
  self->ctx.accumulator=NULL;
  return 0;
}

static int pipeline_alloc_lut(pipeline_t self, unsigned inwidth)
{ if(self->ctx.lut)
  { CUTRY(hipFree(self->ctx.lut));
    CUTRY(hipFree(self->ctx.lut_fpart));
    CUTRY(hipFree(self->ctx.lut_norms));
  }
  CUNEW(unsigned,self->ctx.lut,        inwidth);
  CUNEW(float   ,self->ctx.lut_fpart,  inwidth);
  CUNEW(float   ,self->ctx.lut_norms,2*inwidth);
  return 1;
Error:
  self->ctx.lut=NULL;
  self->ctx.lut_fpart=NULL;
  self->ctx.lut_norms=NULL;
  return 0;

}


static int pipeline_fill_lut(pipeline_t self, unsigned inwidth)
{ int isok=1;
  unsigned * __restrict__ lut=0;
  float    * __restrict__ fpart=0,
           * __restrict__ norms=0,
           * __restrict__ hits=0;
  // useful constants
  const double        d = inwidth/self->samples_per_scan; // 1 - duty
  const double    halfd = d/2.0;
  const unsigned     ow = pipeline_get_output_width(self,inwidth);
  const double        s = (1.0-d)/(double)ow;
  const double     cosd = cos(M_PI*d);
  // alloc temporary space
  NEW(unsigned ,lut  ,inwidth);
  NEW(float    ,fpart,inwidth);
  NEW(float    ,norms,2*inwidth);
  NEW(float    ,hits ,ow+1);
  ZERO(unsigned,lut  ,inwidth);
  ZERO(float   ,fpart,inwidth);
  ZERO(float   ,norms,2*inwidth);
  ZERO(float   ,hits ,ow+1);

  // compute lookup
  for(unsigned i=0;i<inwidth;++i)
  { double phase=halfd+s*i;
    double v=(1.0-cos(2*M_PI*phase)-(1-cosd) )*ow/2.0/cosd;
    unsigned j;
    j = lut[i]   =   floor(v);
    fpart[i]     = v-floor(v);
    hits[j  ]+=fpart[i];
    hits[j+1]+=(1.0f-fpart[i]);
  }
  // compute norms
  for(unsigned i=0;i<inwidth;++i)
  { const unsigned j=lut[i];
    norms[2*i  ]=1.0f/hits[j  ];
    norms[2*i+1]=1.0f/hits[j+1];
  }

  // upload
  CUTRY(hipMemcpy(self->ctx.lut      ,lut  ,  inwidth*sizeof(unsigned),hipMemcpyHostToDevice));
  CUTRY(hipMemcpy(self->ctx.lut_fpart,fpart,  inwidth*sizeof(float)   ,hipMemcpyHostToDevice));
  CUTRY(hipMemcpy(self->ctx.lut_norms,norms,2*inwidth*sizeof(float)   ,hipMemcpyHostToDevice));

Finalize:  
  if(lut)   free(lut);
  if(fpart) free(fpart);
  if(norms) free(norms);
  if(hits)  free(hits);
  return isok;
Error:
  isok=0;
  goto Finalize;
}

static int pipeline_upload(pipeline_t self, pipeline_image_t dst, const pipeline_image_t src)
{ if(self->src && (self->ctx.w>src->w || self->ctx.h>src->h))
  { CUTRY(hipFree(self->src)); self->src=0;
    CUTRY(hipFree(self->dst)); self->dst=0;
    CUTRY(hipFree(self->tmp)); self->tmp=0;
  }
  if(!self->src)
  { CUTRY(hipMalloc((void**)&self->src,pipeline_image_nbytes(src)));
    CUTRY(hipMalloc((void**)&self->dst,pipeline_image_nbytes(dst)));
    CUTRY(hipMalloc((void**)&self->tmp,pipeline_image_nelem(dst)*sizeof(float)));
  }
  CUTRY(hipMemset(self->tmp,0,pipeline_image_nelem(dst)*sizeof(float)));
  CUTRY(hipMemcpy(self->src,src->data,pipeline_image_nbytes(src),hipMemcpyHostToDevice));
  self->ctx.w=src->w;
  self->ctx.h=src->h;
  self->ctx.istride=src->stride;
  self->ctx.ostride=dst->stride;
  return 1;
Error:
  return 0;
}

static int pipeline_download(pipeline_t self, pipeline_image_t dst)
{ TRY(self->dst);
  CUTRY(hipMemcpy(self->dst,dst->data,pipeline_image_nbytes(dst),hipMemcpyDeviceToHost));
  return 1;
Error:
  return 0;
}

int pipeline_exec(pipeline_t self, pipeline_image_t dst, const pipeline_image_t src)
{ if(src->w>self->ctx.w)
  { TRY(pipeline_alloc_accumulator(self,src)); // these will free if one already exists
    TRY(pipeline_alloc_lut(self,src->w));
    TRY(pipeline_fill_lut(self,src->w));
  }
  pipeline_image_conversion_params(dst,src,self->invert,&self->ctx.m,&self->ctx.b);
  TRY(pipeline_upload(self,dst,src)); // updates context size and stride as well
// launch kernel
  TRY(pipeline_download(self,dst));
  return 1;
Error:
  return 0;
}

/* PLAN
  INPUT
  * Count on 32MB/image (4 channels, no downsampling)
  * Expect 32 pixel aligned rows (16 after wrap)
      - one block will access BX*WORK columns

  1. VRAM based accumulator for frame averaging
     Just need the space for one extra image.
     Call to kernel increments accumulator which will write an image
     to the destination buffer every k calls.

     * Accumulator is aligned to BX*WORK
  2. cast and invert are pixel-wise and can be done inline.  frame format is implicit
  3. wrap and unwarp are combined
    1. wrap
       Rows have an even number of pixels so turn will be aligned to a pixel border.  This
       makes this just a copy...might be able to do this in place.
    2. unwrap 
       This is a lut mapping...could introduce linear interp, though this could 
       be taken care of with the lut and the norm
    3. subsampling is done by lut

  THREADS PER BLOCK (BX,BY)
    - BX   should be 1 warp (32)
    - BY   should be set to run max threads per block (BY=1024/BX)
    - WORK should be just enough to hide memory latency
  BLOCKS
    X: ceil( width/BX/WORK )
    Y: ceil( height*nchan/BY)
*/